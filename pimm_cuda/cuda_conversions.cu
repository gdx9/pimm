#include "hip/hip_runtime.h"
#include "pimm_cuda.cuh"
#include <cmath>

using namespace std;

namespace pimm::cuda {
// kernels
__global__
void RgbToGrayMean_kernel(uint8_t* rgb888, uint8_t* gray,
    const size_t kWidth, const size_t kHeight, const size_t kSizeGray){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < kHeight && idx < kWidth) {
        size_t pos = kWidth * idy + idx;

        int32_t val = (static_cast<int32_t>(rgb888[pos+0])
                    + static_cast<int32_t>(rgb888[pos+kSizeGray])
                    + static_cast<int32_t>(rgb888[pos+(kSizeGray<<1)])) / 3;
        // clip
        gray[pos] = static_cast<uint8_t>((val > 255) ? 255 : (val < 0) ? 0 : val);
    }
}
__global__
void RgbToGrayMean_kernel2(uint8_t* rgb888, uint8_t* gray,
    const size_t kSizeGray){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < kSizeGray) {
        int32_t val = (static_cast<int32_t>(rgb888[tid+0])
                    + static_cast<int32_t>(rgb888[tid+kSizeGray])
                    + static_cast<int32_t>(rgb888[tid+(kSizeGray<<1)])) / 3;
        // clip
        gray[tid] = static_cast<uint8_t>((val > 255) ? 255 : (val < 0) ? 0 : val);
    }
}

__global__
void RgbToGrayWeighted_kernel(uint8_t* rgb888, uint8_t* gray,
    const size_t kWidth, const size_t kHeight, const size_t kSizeGray){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < kHeight && idx < kWidth) {
        size_t pos = kWidth * idy + idx;

        float val = (static_cast<float>(rgb888[pos+0]) * 0.299f
                    + static_cast<float>(rgb888[pos+kSizeGray]) * 0.587f
                    + static_cast<float>(rgb888[pos+(kSizeGray<<1)]) * 0.144f);
        // clip
        gray[pos] = static_cast<uint8_t>((val > 255) ? 255 : (val < 0) ? 0 : val);
    }
}

__global__
void InvertColor_kernel(uint8_t* rgb, uint8_t* inverted, const size_t kNumElements){
    // int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < kNumElements) {
        inverted[tid] = 255 - rgb[tid];// r

    }
}

__global__
void SolariseColor_kernel(uint8_t* rgb888, uint8_t* solarize, const size_t kNumElements, const uint8_t kThreshold){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < kNumElements) {
        uint8_t val = rgb888[tid];
        solarize[tid] = (val < kThreshold) ? (255 - val) : val;
    }

}

__global__
void PrepareGammaLutTable_kernel(uint8_t* gamma_lut_table, const float kGamma){
    // prepare gamma lut table

    const size_t kNumElements = 256;
    for(size_t i = 0; i < kNumElements; ++i){
        float val = pow(static_cast<float>(i) / 255.f, kGamma) * 255.f;

        // clip value
        gamma_lut_table[i] = static_cast<uint8_t>(
            (val < 0.f) ? 0.f : (val > 255.f) ? 255.f : val);
    }
}

__global__
void AdjustGamma_kernel(uint8_t* rgb888_src, uint8_t* rgb888_dst, uint8_t* gamma_lut_table, const size_t kNumElements){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < kNumElements) {
        rgb888_dst[tid] = gamma_lut_table[rgb888_src[tid]];
    }
}

__global__
void PrepareContrastLutTable_kernel(uint8_t* contrast_lut_table, const float kContrast){
    //F = 259*(255+contrast)/(255 *(259-contrast))
    const float kF = 259.f * (255.f + kContrast) / (255.f * (259.f - kContrast));

    const size_t kNumElements = 256;

    for(size_t i = 0; i < kNumElements; ++i){
        float val = (kF * static_cast<float>(i - 128.f)) + 128.f;

        // clip value
        contrast_lut_table[i] = static_cast<uint8_t>((val < 0.f) ? 0.f : (val > 255.f) ? 255.f : val);
    }

}

__global__
void AdjustContrast_kernel(uint8_t* rgb888_src, uint8_t* rgb888_dst, uint8_t* contrast_lut_table, const size_t kNumElements){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < kNumElements) {
        rgb888_dst[tid] = contrast_lut_table[rgb888_src[tid]];
    }
}

__global__
void AdjustBrightness_kernel(uint8_t* rgb888_src, uint8_t* rgb888_dst, const size_t kNumElements, const int32_t kBrightness){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < kNumElements) {
        int32_t val = static_cast<int32_t>(rgb888_src[tid]) + kBrightness;
        rgb888_dst[tid] = static_cast<uint8_t>((val < 0) ? 0 : (val > 255) ? 255 : val);
    }
}

extern hipStream_t processing_stream;

void RgbToGrayMean(uint8_t* const rgb888, uint8_t* gray, const size_t kWidth, const size_t kHeight){
    dim3 tpb(32, 8);
    //1200,799
    dim3 nb(kWidth / tpb.x + 1, kHeight / tpb.y + 1);// 'walk' only on half of image - (WIDTH / 2)

    RgbToGrayMean_kernel<<<nb, tpb, 0, processing_stream>>>(rgb888, gray, kWidth, kHeight, kWidth*kHeight);
    hipStreamSynchronize(processing_stream);

    // size_t kSizeGray = kWidth * kHeight;
    // int num_threads = 256;// 32*8
    // int num_blocks = (kNumElements*3 + num_threads - 1) / num_threads;
    // RgbToGrayMean_kernel2<<<num_blocks, num_threads, 0, processing_stream>>>(rgb888, gray, kSizeGray);
    // hipStreamSynchronize(processing_stream);

}

void RgbToGrayWeighted(uint8_t* const rgb888, uint8_t* gray, const size_t kWidth, const size_t kHeight){
    dim3 tpb(32, 8);
    //1200,799
    dim3 nb(kWidth / tpb.x + 1, kHeight / tpb.y + 1);// 'walk' only on half of image - (WIDTH / 2)

    RgbToGrayWeighted_kernel<<<nb, tpb, 0, processing_stream>>>(rgb888, gray, kWidth, kHeight, kWidth*kHeight);
    hipStreamSynchronize(processing_stream);
}

void InvertColor(uint8_t* const rgb888_src, uint8_t* rgb888_dst, const size_t kNumElements){
    int num_threads = 256;// 32*8
    int num_blocks = (kNumElements*3 + num_threads - 1) / num_threads;
	InvertColor_kernel << <num_blocks, num_threads, 0, processing_stream >> > (rgb888_src, rgb888_dst, kNumElements);
    hipStreamSynchronize(processing_stream);
}

void SolariseColor(uint8_t* const rgb888_src, uint8_t* rgb888_dst,
    const size_t kNumElements, const uint8_t kThreshold){
    int num_threads = 256;// 32*8
	int num_blocks = (kNumElements*3 + num_threads - 1) / num_threads;
	SolariseColor_kernel<<<num_blocks, num_threads, 0, processing_stream>>>(rgb888_src, rgb888_dst, kNumElements, kThreshold);
    hipStreamSynchronize(processing_stream);
}

void AdjustGamma(uint8_t* const rgb888_src, uint8_t* rgb888_dst,
    const size_t kNumElements, const float kGamma){
    uint8_t* gamma_lut_table;
    hipMalloc(&gamma_lut_table, 256 * sizeof(uint8_t));

    PrepareGammaLutTable_kernel<<<1, 1, 0, processing_stream>>>(gamma_lut_table, kGamma);

    int num_threads = 256;// 32*8
    int num_blocks = (kNumElements*3 + num_threads - 1) / num_threads;
    AdjustGamma_kernel<<<num_blocks, num_threads, 0, processing_stream>>>(rgb888_src, rgb888_dst, gamma_lut_table, kNumElements);
    hipStreamSynchronize(processing_stream);

    hipFree(gamma_lut_table);
    gamma_lut_table = nullptr;
}

void AdjustContrast(uint8_t* const rgb888_src, uint8_t* rgb888_dst,
    const size_t kNumElements, const float kContrast){

    uint8_t* contrast_lut_table;
    hipMalloc(&contrast_lut_table, 256 * sizeof(uint8_t));

    PrepareContrastLutTable_kernel<<<1, 1, 0, processing_stream>>>(contrast_lut_table, kContrast);

    int num_threads = 256;// 32*8
    int num_blocks = (kNumElements*3 + num_threads - 1) / num_threads;
    AdjustContrast_kernel<<<num_blocks, num_threads, 0, processing_stream>>>(rgb888_src, rgb888_dst, contrast_lut_table, kNumElements);
    hipStreamSynchronize(processing_stream);

    hipFree(contrast_lut_table);
    contrast_lut_table = nullptr;
}

void AdjustBrightness(uint8_t* const rgb888_src, uint8_t* rgb888_dst,
    const size_t kNumElements, const int32_t kBrightness){
    int num_threads = 256;// 32*8
    int num_blocks = (kNumElements*3 + num_threads - 1) / num_threads;
    AdjustBrightness_kernel<<<num_blocks, num_threads, 0, processing_stream>>>(rgb888_src, rgb888_dst, kNumElements, kBrightness);
    hipStreamSynchronize(processing_stream);
}

}
