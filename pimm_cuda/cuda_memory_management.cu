#include "cuda_memory_management.cuh"

using namespace pimm::utils;

namespace pimm::cuda {

hipStream_t processing_stream;

void init(){
    hipStreamCreate(&processing_stream);
}

void close(){
    hipStreamDestroy(processing_stream);
}

uint8_t* AllocateImageMemoryBytes(const size_t kWidth, const size_t kHeight, const COLOR_MODEL kColorModel){
    const size_t kNumElements = GetNumElementsForColorModel(kWidth, kHeight, kColorModel);
    if(kNumElements == 0){
        return nullptr;
    }

    uint8_t* dev_image_bytes;
    hipMalloc(&dev_image_bytes, kNumElements * sizeof(uint8_t));

    return dev_image_bytes;
}

void ReleaseImageMemoryBytes(uint8_t* image_bytes){
    hipFree(image_bytes);
    image_bytes = nullptr;
}

void CopyCpuToGpu(uint8_t* from, uint8_t* to, const size_t kNumBytes){
    //hipMemcpy(to, from, kNumBytes, hipMemcpyHostToDevice);
    hipMemcpyAsync(to, from, kNumBytes, hipMemcpyHostToDevice, processing_stream);
    hipStreamSynchronize(processing_stream);
}

void CopyGpuToGpu(uint8_t* from, uint8_t* to, const size_t kNumBytes){
    //hipMemcpy(to, from, kNumBytes, hipMemcpyHostToDevice);
    hipMemcpyAsync(to, from, kNumBytes, hipMemcpyDeviceToDevice, processing_stream);
    hipStreamSynchronize(processing_stream);
}
void CopyGpuToCpu(uint8_t* from, uint8_t* to, const size_t kNumBytes){
    //hipMemcpy(to, from, kNumBytes, hipMemcpyHostToDevice);
    hipMemcpyAsync(to, from, kNumBytes, hipMemcpyDeviceToHost, processing_stream);
    hipStreamSynchronize(processing_stream);
}

}
